#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <math.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <time.h>
#include <omp.h>
using namespace std;
using namespace cv;

void gaussian(Mat src, Mat dst);
void neighborhood_averaging(Mat src, Mat dst);
void median_flitering(Mat src, Mat dst);
void quickSort(int *a, int left, int right);
void peak_and_valley_flitering(Mat src, Mat dst);
void max_flitering(Mat src, Mat dst);
void min_flitering(Mat src, Mat dst);
void colorchange(Mat src, Mat dst);
double spacedistance(int x1, int y1, int x2, int y2, double sigmaS);
double GSdistance(int g1, int g2, double sigmaG);
void bilateralfilter(Mat src, Mat dst, double sigmaS, double sigmaG);

int main() {
	Mat src = imread("freckle.jpg", CV_LOAD_IMAGE_GRAYSCALE);
	Mat dst(src.rows, src.cols, CV_8U);
	Mat dst2(src.rows, src.cols, CV_8U);
	Mat dst3(src.rows, src.cols, CV_8U);
	Mat dst4(src.rows, src.cols, CV_8U);
	Mat dst5(src.rows, src.cols, CV_8U);

	unsigned long start = clock();
	neighborhood_averaging(src, dst);
	unsigned long end = clock();
	cout << "neigbor total time=" << (end - start) / 1000.0 << "seconds" << endl;

	start = clock();
	gaussian(src, dst2);
	end = clock();
	cout << "guassian total time=" << (end - start) / 1000.0 << "seconds" << endl;

	start = clock();
	median_flitering(src, dst3);
	end = clock();
	cout << "median total time=" << (end - start) / 1000.0 << "seconds" << endl;

	start = clock();
	peak_and_valley_flitering(src, dst4);
	end = clock();
	cout << "peak_and_valley total time=" << (end - start) / 1000.0 << "seconds" << endl;


	start = clock();
	//dim3 blocksPerGrid(512, 1, 1)
	//dim3 threadsPerBlock(512, 1, 1)
	//bilateralfilter << <blocksPerGrid, threadsPerBlock >> > ()
	bilateralfilter(src, dst5, 13, 13);
	end = clock();
	cout << "bilateralfilter total time=" << (end - start) / 1000.0 << "seconds" << endl;

	//�̤j�̤p�o�i�b�̤p�̤j�o�i
	/*max_flitering(src,dst);
	min_flitering(dst,dst2);
	min_flitering(dst2,dst3);
	max_flitering(dst3,dst4);*/


	imshow("orgin", src);
	imshow("neigbor", dst);
	imshow("guassian", dst2);
	imshow("median", dst3);
	imshow("peak_and_valley", dst4);
	imshow("bilateralfilter", dst5);

	imwrite("neigbor.jpg", dst);
	imwrite("guassian.jpg", dst2);
	imwrite("median.jpg", dst3);
	imwrite("peak_and_valley.jpg", dst4);
	imwrite("bilateralfilter.jpg", dst5);


	waitKey(0);
	return(0);
}

//�۾F���������k
void neighborhood_averaging(Mat src, Mat dst) {
	int mask[25] = { 1,1,1,1,1,
					1,1,1,1,1,
					1,1,1,1,1,
					1,1,1,1,1,
					1,1,1,1,1 };
	int divisor = 0;
	int m = 2;
	int rows = src.rows;
	int cols = src.cols;

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			int sum = 0;
			int index = 0;
			for (int y = i - m; y <= i + m; y++)
			{
				for (int x = j - m; x <= j + m; x++)
				{
					if (y < 0 || x < 0 || y >= rows || x >= cols)
					{
						continue;
					}
					divisor = divisor + mask[index];
					sum = sum + src.at<uchar>(y, x) * mask[index++];
				}
			}
			sum = sum / divisor;
			if (sum > 255) { sum = 255; }
			dst.at<uchar>(i, j) = sum;
			divisor = 0;
		}
	}
}

//�������Ƥ�
void gaussian(Mat src, Mat dst) {
	int mask[25] = { 1,2,4,2,1,
					2,4,10,4,2,
					4,10,16,10,4,
					2,4,10,4,2,
					1,2,4,2,1, };
	int m = 2; //5x5
	int rows = src.rows;
	int cols = src.cols;
	int divisor = 0;
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			int sum = 0;
			int index = 0;

			for (int y = i - m; y <= i + m; y++)
			{
				for (int x = j - m; x <= j + m; x++)
				{
					if (y < 0 || x < 0 || y >= rows || x >= cols)
					{
						continue;
					}
					divisor = divisor + mask[index];
					sum = sum + src.at<uchar>(y, x) * mask[index++];
				}
			}
			sum = sum / divisor;
			if (sum > 255) { sum = 255; }
			dst.at<uchar>(i, j) = sum;
			divisor = 0;
		}
	}
}

//�����o�i
void median_flitering(Mat src, Mat dst) {
	int temparray[25]; //5x5
	int m = 2;
	int rows = src.rows;
	int cols = src.cols;

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			int index = 0;
			for (int y = i - m; y <= i + m; y++)
			{
				for (int x = j - m; x <= j + m; x++)
				{
					if (y < 0 || x < 0 || y >= rows || x >= cols)
					{
						continue;
					}
					temparray[index++] = src.at<uchar>(y, x);
				}
			}
			quickSort(temparray, 0, sizeof(temparray) / 4 - 1);
			dst.at<uchar>(i, j) = temparray[sizeof(temparray) / 8 - 1];
		}
	}
}

void quickSort(int *a, int left, int right) {
	int pivot, i, j, temp; //pivot=����I�Ftemp=��Ƥ����Ϊ��Ȧs��
	pivot = a[left]; //��ǭȥ�����Ĥ@�ӼƦr
	i = left; //�����pivot�p���ƭȥ�index
	j = right; //�����pivot�j���ƭȥ�index
	temp = 0;

	if (left < right) {
		while (i < j) {
			i++;
			while (a[i] < pivot) {
				i++;
			}
			while (a[j] > pivot) {
				j--;
			}
			//��줣�ŦX���ȮɡA�洫��̦�m -> ���q�p���ǭȡA�k�q�j���ǭ�
			if (i < j) {
				temp = a[j];
				a[j] = a[i];
				a[i] = temp;
			}
		}
		//�N��ǭȴ���w�g���n�j�p����ɪ������h
		temp = a[j];
		a[j] = a[left];
		a[left] = temp;

		//�Q�λ��j�~��ƧǤj�p����
		quickSort(a, left, j - 1); //�Ƨǥ��q
		quickSort(a, j + 1, right);//�Ƨǥk�q
	}
}

//�i�p�i���o�i
void peak_and_valley_flitering(Mat src, Mat dst) {
	int m = 2; //5x5
	int rows = src.rows;
	int cols = src.cols;

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			int min = 256;
			int max = 0;
			for (int y = i - m; y <= i + m; y++)
			{
				for (int x = j - m; x <= j + m; x++)
				{
					if (y < 0 || x < 0 || y >= rows || x >= cols)
					{
						continue;
					}
					if (x != j && y != i)
					{
						if (src.at<uchar>(y, x) > max)
						{
							max = src.at<uchar>(y, x);
						}
						if (src.at<uchar>(y, x) < min)
						{
							min = src.at<uchar>(y, x);
						}
					}


				}
			}
			if (src.at<uchar>(i, j) <= min)
			{
				dst.at<uchar>(i, j) = min;
			}
			else if (src.at<uchar>(i, j) >= max)
			{
				dst.at<uchar>(i, j) = max;
			}
			else {
				dst.at<uchar>(i, j) = src.at<uchar>(i, j);
			}
		}
	}
}

//�̤j�o�i
void max_flitering(Mat src, Mat dst) {
	int m = 1; //3x3
	int rows = src.rows;
	int cols = src.cols;

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			int max = 0;
			for (int y = i - m; y <= i + m; y++)
			{
				for (int x = j - m; x <= j + m; x++)
				{
					if (y < 0 || x < 0 || y >= rows || x >= cols)
					{
						continue;
					}
					if (src.at<uchar>(y, x) > max)
					{
						max = src.at<uchar>(y, x);
					}
				}
			}
			dst.at<uchar>(i, j) = max;
		}
	}
}

//�̤p�o�i
void min_flitering(Mat src, Mat dst) {
	int m = 1; //3x3
	int rows = src.rows;
	int cols = src.cols * src.channels();

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			int min = 256;
			for (int y = i - m; y <= i + m; y++)
			{
				for (int x = j - m; x <= j + m; x++)
				{
					if (y < 0 || x < 0 || y >= rows || x >= cols)
					{
						continue;
					}
					if (src.at<uchar>(y, x) < min)
					{
						min = src.at<uchar>(y, x);
					}
				}
			}
			dst.at<uchar>(i, j) = min;
		}
	}
}

//�Ŷ��Z���t��
//��ӹ��������Z��
double spacedistance(int x1, int y1, int x2, int y2, double sigmaS) {
	double X = pow(abs(x1 - x2), 2);
	double Y = pow(abs(y1 - y2), 2);

	return exp(-(X + Y) / (2 * pow(sigmaS, 2)));
}

//�Ƕ��Z���t��
//�ھڨ�ۦ��{�� ��ӹ������Ȥ������Z��
double GSdistance(int g1, int g2, double sigmaG) {
	double X = pow(abs(g1 - g2), 2);
	return exp(-X / (2 * pow(sigmaG, 2)));
}

//�����o�i��
// sigmaS = sigmaG = 13
void bilateralfilter(Mat src, Mat dst, double sigmaS, double sigmaG) { //�����o�i��
	int m = 7; //15*15
	int rows = src.rows;
	int cols = src.cols;

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			//int i = blockIdx.x * blockDim.x + threadIdx.x;
			//int j = blockIdx.y * blockDim.y + threadIdx.y;
			double k = 0;
			double f = 0;
			for (int y = i - m; y <= i + m; y++)
			{
				for (int x = j - m; x <= j + m; x++)
				{
					if (y < 0 || x < 0 || y >= rows || x >= cols)
					{
						continue;
					}
					// src.at<uchar>(column, row) �]�� MAT �O column-major
					// CV_8U �Ω�8��1�q�D�ǫ׹Ϲ�
					f = f + src.at<uchar>(y, x) * spacedistance(i, j, y, x, sigmaS) * GSdistance(src.at<uchar>(i, j), src.at<uchar>(y, x), sigmaG);
					k = k + spacedistance(i, j, y, x, sigmaS) * GSdistance(src.at<uchar>(i, j), src.at<uchar>(y, x), sigmaG);

				}
			}
			int g = f / k;
			if (g < 0) g = 0;
			else if (g > 255) g = 255;
			dst.at<uchar>(i, j) = (uchar)g;
		}
	}
}
